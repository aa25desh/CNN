#include <iostream>
#include <hip/hip_runtime.h>
//you can change the grid_size
#define GRID_SIZE 128
//you can change the block_size
#define BLOCK_SIZE 128


#define TILE_WIDTH 8

__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){

           int n = blockIdx.x;
           int k = blockIdx.z;
           int p = (blockIdx.y / (P/TILE_WIDTH + 1) )*TILE_WIDTH + threadIdx.y;
           int q = (blockIdx.y % (P/TILE_WIDTH + 1) )*TILE_WIDTH + threadIdx.x;
            
          //for(unsigned int n=0; n<N; n++) { // minibatch size
          //  for(unsigned int k=0; k<K; k ++) { // output feature map
            if (n < N && k < K && p < P && q < Q)
            {// input feature map
                float sum1 = 0;
                unsigned int ij1 = p * u; // input height
                unsigned int ii1 = q * v; // input width
                for(unsigned int c=0; c<C; c ++) { 
                    for (unsigned int r = 0; r<R; r ++) { // filter height
                        for (unsigned int s = 0; s < S; s ++) {// filter width
                            sum1 += d_input[n*C*H*W + c*H*W + (ij1+r)*W + ii1+s] * d_weight[k*C*R*S+c*R*S+r*S+s];
                            }
                        }
                    }
                d_output[n*K*P*Q + k*P*Q + p*Q + q] = sum1;
            }

}
/*
//N = 128, C = 832, K = 128, H = 7, W = 7, R = 1, S = 1, u = 1, v = 1, P = 7, Q = 7 
#define TILE_WIDTH 8
__global__ void cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){

           int n = blockIdx.x;
           int k = blockIdx.z;
           int p = (blockIdx.y / (P/TILE_WIDTH + 1) )*TILE_WIDTH + threadIdx.y;
           int q = (blockIdx.y % (P/TILE_WIDTH + 1) )*TILE_WIDTH + threadIdx.x;
            
          //for(unsigned int n=0; n<N; n++) { // minibatch size
          //  for(unsigned int k=0; k<K; k ++) { // output feature map
            if (n < N && k < K && p < P && q < Q)
            {// input feature map
                float sum1 = 0;
                unsigned int ij1 = p * u; // input height
                unsigned int ii1 = q * v; // input width
                for(unsigned int c=0; c<C; c=c+32) { 
                    //for (unsigned int r = 0; r<R; r ++) { // filter height
                    //    for (unsigned int s = 0; s < S; s ++) {// filter width
                        int _i = n*C*H*W + (ij1)*W + ii1;
                        int _w = k*C*R*S;
                        sum1 += d_input[ _i+ c*H*W] * d_weight[_w+c*R*S];
                        sum1 += d_input[ _i+ (c+1)*H*W] * d_weight[_w+(c+1)*R*S];
                        sum1 += d_input[ _i+ (c+2)*H*W] * d_weight[_w+(c+2)*R*S];
                        sum1 += d_input[ _i+ (c+3)*H*W] * d_weight[_w+(c+3)*R*S];
                        sum1 += d_input[ _i+ (c+4)*H*W] * d_weight[_w+(c+4)*R*S];
                        sum1 += d_input[ _i+ (c+5)*H*W] * d_weight[_w+(c+5)*R*S];
                        sum1 += d_input[ _i+ (c+6)*H*W] * d_weight[_w+(c+6)*R*S];
                        sum1 += d_input[ _i+ (c+7)*H*W] * d_weight[_w+(c+7)*R*S];
                        sum1 += d_input[ _i+ (c+8)*H*W] * d_weight[_w+(c+8)*R*S];
                        sum1 += d_input[ _i+ (c+9)*H*W] * d_weight[_w+(c+9)*R*S];
                        sum1 += d_input[ _i+ (c+10)*H*W] * d_weight[_w+(c+10)*R*S];
                        sum1 += d_input[ _i+ (c+11)*H*W] * d_weight[_w+(c+11)*R*S];
                        sum1 += d_input[ _i+ (c+12)*H*W] * d_weight[_w+(c+12)*R*S];
                        sum1 += d_input[ _i+ (c+13)*H*W] * d_weight[_w+(c+13)*R*S];
                        sum1 += d_input[ _i+ (c+14)*H*W] * d_weight[_w+(c+14)*R*S];
                        sum1 += d_input[ _i+ (c+15)*H*W] * d_weight[_w+(c+15)*R*S];
                        sum1 += d_input[ _i+ (c+16)*H*W] * d_weight[_w+(c+16)*R*S];
                        sum1 += d_input[ _i+ (c+17)*H*W] * d_weight[_w+(c+17)*R*S];
                        sum1 += d_input[ _i+ (c+18)*H*W] * d_weight[_w+(c+18)*R*S];
                        sum1 += d_input[ _i+ (c+19)*H*W] * d_weight[_w+(c+19)*R*S];
                        sum1 += d_input[ _i+ (c+20)*H*W] * d_weight[_w+(c+20)*R*S];
                        sum1 += d_input[ _i+ (c+21)*H*W] * d_weight[_w+(c+21)*R*S];
                        sum1 += d_input[ _i+ (c+22)*H*W] * d_weight[_w+(c+22)*R*S];
                        sum1 += d_input[ _i+ (c+23)*H*W] * d_weight[_w+(c+23)*R*S];
                        sum1 += d_input[ _i+ (c+24)*H*W] * d_weight[_w+(c+24)*R*S];
                        sum1 += d_input[ _i+ (c+25)*H*W] * d_weight[_w+(c+25)*R*S];
                        sum1 += d_input[ _i+ (c+26)*H*W] * d_weight[_w+(c+26)*R*S];
                        sum1 += d_input[ _i+ (c+27)*H*W] * d_weight[_w+(c+27)*R*S];
                        sum1 += d_input[ _i+ (c+28)*H*W] * d_weight[_w+(c+28)*R*S];
                        sum1 += d_input[ _i+ (c+29)*H*W] * d_weight[_w+(c+29)*R*S];
                        sum1 += d_input[ _i+ (c+30)*H*W] * d_weight[_w+(c+30)*R*S];
                        sum1 += d_input[ _i+ (c+31)*H*W] * d_weight[_w+(c+31)*R*S];


                            //}

                }
                        //}
                    //}
                d_output[n*K*P*Q + k*P*Q + p*Q + q] = sum1;
            }

}
/*
//N = 128, C = 3, K = 64, H = 112, W = 112, R = 3, S = 3, u = 2, v = 2, P = 55, Q = 55 
#define TILE_WIDTH 32
__global__ void unroll2_cnn(int N,int C,int K,int H,int W,int R, int S, int u, int v, int P, int Q,
               float *d_input, float * d_weight, float * d_output){

           int n = blockIdx.x;
           int k = blockIdx.z;
           int p = (blockIdx.y / (P/TILE_WIDTH + 1) )*TILE_WIDTH + threadIdx.y;
           int q = (blockIdx.y % (P/TILE_WIDTH + 1) )*TILE_WIDTH + threadIdx.x;
            
          //for(unsigned int n=0; n<N; n++) { // minibatch size
          //  for(unsigned int k=0; k<K; k ++) { // output feature map
            if (n < N && k < K && p < P && q < Q)
            {// input feature map
                float sum1 = 0;
                //for(unsigned int c=0; c<C; c ++) { 
                    //for (unsigned int r = 0; r<R; r ++) { // filter height
                    //    for (unsigned int s = 0; s < S; s ++) {// filter width
                            int _i = n*C*H*W + p*u*W + q*v;
                            int _w = k*C*R*S;
                            sum1 = sum1 + d_input[_i + 0*H*W + 0*W +0] * d_weight[_w+0*R*S+0*S+0] + 
                                          d_input[_i + 0*H*W + 0*W +1] * d_weight[_w+0*R*S+0*S+1] + 
                                          d_input[_i + 0*H*W + 0*W +2] * d_weight[_w+0*R*S+0*S+2] +
                                          d_input[_i + 0*H*W + 1*W +0] * d_weight[_w+0*R*S+1*S+0] + 
                                          d_input[_i + 0*H*W + 1*W +1] * d_weight[_w+0*R*S+1*S+1] + 
                                          d_input[_i + 0*H*W + 1*W +2] * d_weight[_w+0*R*S+1*S+2] +
                                          d_input[_i + 0*H*W + 2*W +0] * d_weight[_w+0*R*S+2*S+0] + 
                                          d_input[_i + 0*H*W + 2*W +1] * d_weight[_w+0*R*S+2*S+1] + 
                                          d_input[_i + 0*H*W + 2*W +2] * d_weight[_w+0*R*S+2*S+2] +
                                          d_input[_i + 1*H*W + 0*W +0] * d_weight[_w+1*R*S+0*S+0] +
                                            d_input[_i + 1*H*W + 0*W +1] * d_weight[_w+1*R*S+0*S+1] +
                                            d_input[_i + 1*H*W + 0*W +2] * d_weight[_w+1*R*S+0*S+2] +
                                            d_input[_i + 1*H*W + 1*W +0] * d_weight[_w+1*R*S+1*S+0] +
                                            d_input[_i + 1*H*W + 1*W +1] * d_weight[_w+1*R*S+1*S+1] +
                                            d_input[_i + 1*H*W + 1*W +2] * d_weight[_w+1*R*S+1*S+2] +
                                            d_input[_i + 1*H*W + 2*W +0] * d_weight[_w+1*R*S+2*S+0] +
                                            d_input[_i + 1*H*W + 2*W +1] * d_weight[_w+1*R*S+2*S+1] +
                                            d_input[_i + 1*H*W + 2*W +2] * d_weight[_w+1*R*S+2*S+2] +
                                            d_input[_i + 2*H*W + 0*W +0] * d_weight[_w+2*R*S+0*S+0] +
                                            d_input[_i + 2*H*W + 0*W +1] * d_weight[_w+2*R*S+0*S+1] +
                                            d_input[_i + 2*H*W + 0*W +2] * d_weight[_w+2*R*S+0*S+2] +
                                            d_input[_i + 2*H*W + 1*W +0] * d_weight[_w+2*R*S+1*S+0] +
                                            d_input[_i + 2*H*W + 1*W +1] * d_weight[_w+2*R*S+1*S+1] +
                                            d_input[_i + 2*H*W + 1*W +2] * d_weight[_w+2*R*S+1*S+2] +
                                            d_input[_i + 2*H*W + 2*W +0] * d_weight[_w+2*R*S+2*S+0] +
                                            d_input[_i + 2*H*W + 2*W +1] * d_weight[_w+2*R*S+2*S+1] +
                                            d_input[_i + 2*H*W + 2*W +2] * d_weight[_w+2*R*S+2*S+2];


                            //}
                        //}
                   // }
                d_output[n*K*P*Q + k*P*Q + p*Q + q] = sum1;
            }

}

*/

